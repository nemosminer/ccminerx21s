#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include "miner.h"

#include <stdio.h>
#include <stdint.h>
#include <memory.h>

#define SPH_C64(x)    ((uint64_t)(x ## ULL))
#define SPH_C32(x)    ((uint32_t)(x ## U))
#define SPH_T32(x)    (x)
#define ROTR(x, n)    (((x) >> (n)) | ((x) << (32 - (n))))
#include "cuda_helper.h"


static __constant__ const uint32_t H256[8] = {
	SPH_C32(0x6A09E667), SPH_C32(0xBB67AE85), SPH_C32(0x3C6EF372),
	SPH_C32(0xA54FF53A), SPH_C32(0x510E527F), SPH_C32(0x9B05688C),
	SPH_C32(0x1F83D9AB), SPH_C32(0x5BE0CD19)
};


__device__ __forceinline__
uint32_t Maj(const uint32_t a, const uint32_t b, const uint32_t c) { //Sha256 - Maj - andor
	uint32_t result;
	asm ("lop3.b32 %0, %1, %2, %3, 0xE8;" : "=r"(result) : "r"(a), "r"(b),"r"(c)); // 0xE8 = ((0xF0 & (0xCC | 0xAA)) | (0xCC & 0xAA))
	return result;
}

#define MAJ(X, Y, Z)   (((X) & (Y)) | (((X) | (Y)) & (Z)))


static __device__ __forceinline__ void sha2_step1(uint32_t a,uint32_t b,uint32_t c,uint32_t &d,uint32_t e,uint32_t f,uint32_t g,uint32_t &h,
	                                              uint32_t in,const uint32_t Kshared) {
	uint32_t t1,t2;
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 =ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
	uint32_t bsg20 =ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
	uint32_t andorv = Maj(a, b, c);		//((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

	t1 = h + bsg21 + vxandx + Kshared + in;
	t2 = bsg20 + andorv;
	d = d + t1;
	h = t1 + t2;
}

static __device__ __forceinline__ void sha2_step2(uint32_t a,uint32_t b,uint32_t c,uint32_t &d,uint32_t e,uint32_t f,uint32_t g,uint32_t &h,
	                                              uint32_t* in,uint32_t pc,const uint32_t Kshared) {
	uint32_t t1,t2;

	int pcidx1 = (pc-2) & 0xF;
	int pcidx2 = (pc-7) & 0xF;
	int pcidx3 = (pc-15) & 0xF;
	uint32_t inx0 = in[pc];
	uint32_t inx1 = in[pcidx1];
	uint32_t inx2 = in[pcidx2];
	uint32_t inx3 = in[pcidx3];


	uint32_t ssg21 = ROTR(inx1, 17) ^ ROTR(inx1, 19) ^ SPH_T32((inx1) >> 10); //ssg2_1(inx1);
	uint32_t ssg20 = ROTR(inx3, 7) ^ ROTR(inx3, 18) ^ SPH_T32((inx3) >> 3); //ssg2_0(inx3);
	uint32_t vxandx = (((f) ^ (g)) & (e)) ^ (g); // xandx(e, f, g);
	uint32_t bsg21 =ROTR(e, 6) ^ ROTR(e, 11) ^ ROTR(e, 25); // bsg2_1(e);
	uint32_t bsg20 =ROTR(a, 2) ^ ROTR(a, 13) ^ ROTR(a, 22); //bsg2_0(a);
	uint32_t andorv = Maj(a, b, c);		//((b) & (c)) | (((b) | (c)) & (a)); //andor32(a,b,c);

	in[pc] = ssg21+inx2+ssg20+inx0;

	t1 = h + bsg21 + vxandx + Kshared + in[pc];
	t2 = bsg20 + andorv;
	d =  d + t1;
	h = t1 + t2;
}


static __device__ __forceinline__ void sha2_round_body(uint32_t* in, uint32_t* r) {
	uint32_t a = r[0];
	uint32_t b = r[1];
	uint32_t c = r[2];
	uint32_t d = r[3];
	uint32_t e = r[4];
	uint32_t f = r[5];
	uint32_t g = r[6];
	uint32_t h = r[7];

	sha2_step1(a,b,c,d,e,f,g,h,in[ 0],0x428A2F98);
	sha2_step1(h,a,b,c,d,e,f,g,in[ 1],0x71374491);
	sha2_step1(g,h,a,b,c,d,e,f,in[ 2],0xB5C0FBCF);
	sha2_step1(f,g,h,a,b,c,d,e,in[ 3],0xE9B5DBA5);
	sha2_step1(e,f,g,h,a,b,c,d,in[ 4],0x3956C25B);
	sha2_step1(d,e,f,g,h,a,b,c,in[ 5],0x59F111F1);
	sha2_step1(c,d,e,f,g,h,a,b,in[ 6],0x923F82A4);
	sha2_step1(b,c,d,e,f,g,h,a,in[ 7],0xAB1C5ED5);
	sha2_step1(a,b,c,d,e,f,g,h,in[ 8],0xD807AA98);
	sha2_step1(h,a,b,c,d,e,f,g,in[ 9],0x12835B01);
	sha2_step1(g,h,a,b,c,d,e,f,in[10],0x243185BE);
	sha2_step1(f,g,h,a,b,c,d,e,in[11],0x550C7DC3);
	sha2_step1(e,f,g,h,a,b,c,d,in[12],0x72BE5D74);
	sha2_step1(d,e,f,g,h,a,b,c,in[13],0x80DEB1FE);
	sha2_step1(c,d,e,f,g,h,a,b,in[14],0x9BDC06A7);
	sha2_step1(b,c,d,e,f,g,h,a,in[15],0xC19BF174);

	sha2_step2(a,b,c,d,e,f,g,h,in, 0,0xE49B69C1);
	sha2_step2(h,a,b,c,d,e,f,g,in, 1,0xEFBE4786);
	sha2_step2(g,h,a,b,c,d,e,f,in, 2,0x0FC19DC6);
	sha2_step2(f,g,h,a,b,c,d,e,in, 3,0x240CA1CC);
	sha2_step2(e,f,g,h,a,b,c,d,in, 4,0x2DE92C6F);
	sha2_step2(d,e,f,g,h,a,b,c,in, 5,0x4A7484AA);
	sha2_step2(c,d,e,f,g,h,a,b,in, 6,0x5CB0A9DC);
	sha2_step2(b,c,d,e,f,g,h,a,in, 7,0x76F988DA);
	sha2_step2(a,b,c,d,e,f,g,h,in, 8,0x983E5152);
	sha2_step2(h,a,b,c,d,e,f,g,in, 9,0xA831C66D);
	sha2_step2(g,h,a,b,c,d,e,f,in,10,0xB00327C8);
	sha2_step2(f,g,h,a,b,c,d,e,in,11,0xBF597FC7);
	sha2_step2(e,f,g,h,a,b,c,d,in,12,0xC6E00BF3);
	sha2_step2(d,e,f,g,h,a,b,c,in,13,0xD5A79147);
	sha2_step2(c,d,e,f,g,h,a,b,in,14,0x06CA6351);
	sha2_step2(b,c,d,e,f,g,h,a,in,15,0x14292967);

	sha2_step2(a,b,c,d,e,f,g,h,in, 0,0x27B70A85);
	sha2_step2(h,a,b,c,d,e,f,g,in, 1,0x2E1B2138);
	sha2_step2(g,h,a,b,c,d,e,f,in, 2,0x4D2C6DFC);
	sha2_step2(f,g,h,a,b,c,d,e,in, 3,0x53380D13);
	sha2_step2(e,f,g,h,a,b,c,d,in, 4,0x650A7354);
	sha2_step2(d,e,f,g,h,a,b,c,in, 5,0x766A0ABB);
	sha2_step2(c,d,e,f,g,h,a,b,in, 6,0x81C2C92E);
	sha2_step2(b,c,d,e,f,g,h,a,in, 7,0x92722C85);
	sha2_step2(a,b,c,d,e,f,g,h,in, 8,0xA2BFE8A1);
	sha2_step2(h,a,b,c,d,e,f,g,in, 9,0xA81A664B);
	sha2_step2(g,h,a,b,c,d,e,f,in,10,0xC24B8B70);
	sha2_step2(f,g,h,a,b,c,d,e,in,11,0xC76C51A3);
	sha2_step2(e,f,g,h,a,b,c,d,in,12,0xD192E819);
	sha2_step2(d,e,f,g,h,a,b,c,in,13,0xD6990624);
	sha2_step2(c,d,e,f,g,h,a,b,in,14,0xF40E3585);
	sha2_step2(b,c,d,e,f,g,h,a,in,15,0x106AA070);

	sha2_step2(a,b,c,d,e,f,g,h,in, 0,0x19A4C116);
	sha2_step2(h,a,b,c,d,e,f,g,in, 1,0x1E376C08);
	sha2_step2(g,h,a,b,c,d,e,f,in, 2,0x2748774C);
	sha2_step2(f,g,h,a,b,c,d,e,in, 3,0x34B0BCB5);
	sha2_step2(e,f,g,h,a,b,c,d,in, 4,0x391C0CB3);
	sha2_step2(d,e,f,g,h,a,b,c,in, 5,0x4ED8AA4A);
	sha2_step2(c,d,e,f,g,h,a,b,in, 6,0x5B9CCA4F);
	sha2_step2(b,c,d,e,f,g,h,a,in, 7,0x682E6FF3);
	sha2_step2(a,b,c,d,e,f,g,h,in, 8,0x748F82EE);
	sha2_step2(h,a,b,c,d,e,f,g,in, 9,0x78A5636F);
	sha2_step2(g,h,a,b,c,d,e,f,in,10,0x84C87814);
	sha2_step2(f,g,h,a,b,c,d,e,in,11,0x8CC70208);
	sha2_step2(e,f,g,h,a,b,c,d,in,12,0x90BEFFFA);
	sha2_step2(d,e,f,g,h,a,b,c,in,13,0xA4506CEB);
	sha2_step2(c,d,e,f,g,h,a,b,in,14,0xBEF9A3F7);
	sha2_step2(b,c,d,e,f,g,h,a,in,15,0xC67178F2);

	r[0] = r[0] + a;
	r[1] = r[1] + b;
	r[2] = r[2] + c;
	r[3] = r[3] + d;
	r[4] = r[4] + e;
	r[5] = r[5] + f;
	r[6] = r[6] + g;
	r[7] = r[7] + h;
}


__global__ void __launch_bounds__(512,2) sha256_gpu_hash_64(int threads, uint32_t *g_hash)
{
	int thread = (blockDim.x * blockIdx.x + threadIdx.x);
	if (thread < threads) {
    uint32_t in[16], in2[16], buf[8];
    uint32_t* inout = &g_hash[thread<<4];

    #pragma unroll
		for (int i = 0; i < 8; i++) buf[i] = H256[i];

		#pragma unroll
		for (int i = 0; i < 16; i++) in[i] = cuda_swab32(inout[i]);
		sha2_round_body(in,buf);

		in2[0] = 0x80000000;
		#pragma unroll
		for (int i = 1 ; i < 15; i++) in2[i] = 0;
		in2[15] = 0x200;
		sha2_round_body(in2,buf);

		#pragma unroll
		for (int i = 0; i < 8; i++) inout[i] = cuda_swab32(buf[i]);
	}
}


__host__
void sha256_cpu_hash_64(int thr_id, int threads, uint32_t *d_hash) {
	const int threadsperblock = 512;
	dim3 grid(threads/threadsperblock);
	dim3 block(threadsperblock);
	sha256_gpu_hash_64<<<grid, block>>>(threads, d_hash);
}
